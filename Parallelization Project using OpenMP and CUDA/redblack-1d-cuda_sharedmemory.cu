/*
 * redblack-1d.c: U. Kremer for cs515, April 2016
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define DATA_TYPE double
#define TIME_STEPS 1000
#define N 750000
#define RADIUS 1
#define THREAD_NUM 8
#define BLOCKSIZE 8

/* Array initialization. */
static
void init_array (DATA_TYPE *a)
{
  int i;

  for (i = 0; i < N; i++) {
    a[i] = (DATA_TYPE) random() /  N; 
  }
}


/* Must scan some live-out data so compiler cannot 
   use dead code elimination (DCE).
   Can be used also to check the correctness of the output. */
static
void print_array_elem(int i, DATA_TYPE *a)
{
  if ((0 <= i) && (i <= N))  
    fprintf(stderr, "(%d):%0.2lf\n", i, a[i]);
  else
    fprintf(stderr, "Error: out of bounds access \n");
}

static
void print_array(DATA_TYPE *a)
{
  int i;

  for (i = 0; i < N; i++) {
    fprintf(stderr, "(%d):%0.2lf  ", i, a[i]);
    if ((i+1)% 5 == 0) fprintf(stderr, "\n");
  }
  fprintf(stderr, "\n");
}

__global__ void redblack_odd(DATA_TYPE *a)
{
__shared__ DATA_TYPE temp[THREAD_NUM + 2*RADIUS];

int gindex=threadIdx.x + blockIdx.x * blockDim.x;
int lindex=2*threadIdx.x+RADIUS;

temp[lindex]= a[2*gindex-1];
temp[lindex-RADIUS]=a[2*gindex-2];
temp[lindex+RADIUS]=a[2*gindex];

__syncthreads();

a[2*gindex-1] = 0.25 * (temp[lindex-RADIUS] + 2* temp[lindex] + temp[lindex+RADIUS]);
//a[2*gindex]=lindex+1005;

}

__global__ void redblack_even(DATA_TYPE *a)
{
__shared__ DATA_TYPE temp[BLOCKSIZE + 2*RADIUS];

int gindex=threadIdx.x + blockIdx.x * blockDim.x;
int lindex=2*threadIdx.x+RADIUS;

temp[lindex]= a[2*gindex];
temp[lindex-RADIUS]=a[2*gindex-1];
temp[lindex+RADIUS]=a[2*gindex+1];

__syncthreads();


a[2*gindex] = 0.25 * (temp[lindex-RADIUS] + 2* temp[lindex] + temp[lindex+RADIUS]);



}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_redblack_1d(int tsteps,
			DATA_TYPE *a)
{
  int t, i;
DATA_TYPE *d_a;

int blocknum=(N/2+THREAD_NUM-1)/THREAD_NUM;
  int nobytes = N*sizeof(DATA_TYPE);

  for (t = 0; t < TIME_STEPS; t++) {
      hipMalloc((void **)&d_a,nobytes);
	hipMemcpy(d_a,a,nobytes,hipMemcpyHostToDevice);
	redblack_odd<<<blocknum,THREAD_NUM>>>(d_a);
      hipMemcpy(a,d_a,nobytes,hipMemcpyDeviceToHost);
	hipMemcpy(d_a,a,nobytes,hipMemcpyHostToDevice);
	redblack_even<<<blocknum,THREAD_NUM>>>(d_a);
      hipMemcpy(a,d_a,nobytes,hipMemcpyDeviceToHost);
      hipFree(d_a);     
  }
}



int main(void)
{
  int tsteps = TIME_STEPS;
  
  /* Timing */
  double start, end, diff;

  /* Variable declaration/allocation. */
  int nbytes = N*sizeof(DATA_TYPE);
  DATA_TYPE *a_h;

  a_h = (DATA_TYPE *) malloc( nbytes );
  
  /* Initialize array(s). */
  
  init_array (a_h);
  print_array_elem(N/2, a_h);
  print_array_elem(N/3, a_h);
  print_array_elem(N/4, a_h);
  print_array_elem(N/5, a_h);
  print_array_elem(N/6, a_h);
  
  /* Start timer. */
  start = clock();

  /* Run kernel. */
  kernel_redblack_1d (tsteps, a_h); 

  /* Stop timer and print */
  end = clock();
  diff = (end - start) / 1000000;
  printf("Execution time (CPU only): %f\n", diff);

  /* debugging and/or trick the dead code eliminator */
  /* print_array(a_h); */
  print_array_elem(N/2, a_h);
  print_array_elem(N/3, a_h);
  print_array_elem(N/4, a_h);
  print_array_elem(N/5, a_h);
  print_array_elem(N/6, a_h);

  free(a_h);

  return 0;
}
