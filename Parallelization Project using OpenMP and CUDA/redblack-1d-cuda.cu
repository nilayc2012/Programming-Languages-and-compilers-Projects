/*
 * redblack-1d.c: U. Kremer for cs515, April 2016
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define DATA_TYPE double
#define TIME_STEPS 1000
#define N 750000
#define threadnum 8

/* Array initialization. */
static
void init_array (DATA_TYPE *a)
{
  int i;

  for (i = 0; i < N; i++) {
    a[i] = (DATA_TYPE) random() /  N; 
  }
}


/* Must scan some live-out data so compiler cannot 
   use dead code elimination (DCE).
   Can be used also to check the correctness of the output. */
static
void print_array_elem(int i, DATA_TYPE *a)
{
  if ((0 <= i) && (i <= N))  
    fprintf(stderr, "(%d):%0.2lf\n", i, a[i]);
  else
    fprintf(stderr, "Error: out of bounds access \n");
}

static
void print_array(DATA_TYPE *a)
{
  int i;

  for (i = 0; i < N; i++) {
    fprintf(stderr, "(%d):%0.2lf  ", i, a[i]);
    if ((i+1)% 5 == 0) fprintf(stderr, "\n");
  }
  fprintf(stderr, "\n");
}

__global__ void redblack_odd(DATA_TYPE *a)
{
int index=threadIdx.x + blockIdx.x * blockDim.x;
a[2*index-1] = 0.25 * (a[2*index-2] + 2* a[2*index-1] + a[2*index]);

}

__global__ void redblack_even(DATA_TYPE *a)
{
int index=threadIdx.x + blockIdx.x * blockDim.x;

a[2*index] = 0.25 * (a[2*index-1] + 2* a[2*index] + a[2*index+1]);
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_redblack_1d(int tsteps,
			DATA_TYPE *a)
{
  int t, i;
DATA_TYPE *d_a;
int blocknum=(N/2+threadnum-1)/threadnum;
  int nobytes = N*sizeof(DATA_TYPE);

  for (t = 0; t < TIME_STEPS; t++) {

      hipMalloc((void **)&d_a,nobytes);
	hipMemcpy(d_a,a,nobytes,hipMemcpyHostToDevice);
	redblack_odd<<<blocknum,threadnum>>>(d_a);
      hipMemcpy(a,d_a,nobytes,hipMemcpyDeviceToHost);

	hipMemcpy(d_a,a,nobytes,hipMemcpyHostToDevice);
	redblack_even<<<blocknum,threadnum>>>(d_a);
      hipMemcpy(a,d_a,nobytes,hipMemcpyDeviceToHost);
      hipFree(d_a);
      
  }
}



int main(void)
{
  int tsteps = TIME_STEPS;
  
  /* Timing */
  double start, end, diff;

  /* Variable declaration/allocation. */
  int nbytes = N*sizeof(DATA_TYPE);
  DATA_TYPE *a_h;

  a_h = (DATA_TYPE *) malloc( nbytes );
  
  /* Initialize array(s). */
  
  init_array (a_h);
  print_array_elem(N/2, a_h);
  print_array_elem(N/3, a_h);
  print_array_elem(N/4, a_h);
  print_array_elem(N/5, a_h);
  print_array_elem(N/6, a_h);
  
  /* Start timer. */
  start = clock();

  /* Run kernel. */
  kernel_redblack_1d (tsteps, a_h); 

  /* Stop timer and print */
  end = clock();
  diff = (end - start) / 1000000;
  printf("Execution time (CPU only): %f\n", diff);

  /* debugging and/or trick the dead code eliminator */
  /* print_array(a_h); */
  print_array_elem(N/2, a_h);
  print_array_elem(N/3, a_h);
  print_array_elem(N/4, a_h);
  print_array_elem(N/5, a_h);
  print_array_elem(N/6, a_h);

  free(a_h);

  return 0;
}
